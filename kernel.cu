#include "hip/hip_runtime.h"
﻿#include <iostream>
#include <hip/hip_runtime.h>
#include <chrono>
#include <cmath>
#include <omp.h>

#define N 15  // Cambia per 1 milione o 10 milioni

// Costanti host
const float scale_host = 1.0005f;
const float Tx_host = 100.0f;
const float Ty_host = -50.0f;
const float Tz_host = 300.0f;

const float R_host[9] = {
    0.866f, -0.433f,  0.25f,
    0.5f,    0.75f,  -0.433f,
    0.0f,    0.5f,   0.866f
};

// Costanti device
__constant__ float scale = 1.0005f;
__constant__ float Tx = 100.0f;
__constant__ float Ty = -50.0f;
__constant__ float Tz = 300.0f;

__constant__ float R[9] = {
    0.866f, -0.433f,  0.25f,
    0.5f,    0.75f,  -0.433f,
    0.0f,    0.5f,   0.866f
};

__global__ void transformGPU(float* x, float* y, float* z, float* x_p, float* y_p, float* z_p) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N) {
        x_p[i] = scale * (R[0] * x[i] + R[1] * y[i] + R[2] * z[i]) + Tx;
        y_p[i] = scale * (R[3] * x[i] + R[4] * y[i] + R[5] * z[i]) + Ty;
        z_p[i] = scale * (R[6] * x[i] + R[7] * y[i] + R[8] * z[i]) + Tz;
    }
}

void transformCPU_parallel(float* x, float* y, float* z, float* x_p, float* y_p, float* z_p) {
#pragma omp parallel for
    for (int i = 0; i < N; ++i) {
        x_p[i] = scale_host * (R_host[0] * x[i] + R_host[1] * y[i] + R_host[2] * z[i]) + Tx_host;
        y_p[i] = scale_host * (R_host[3] * x[i] + R_host[4] * y[i] + R_host[5] * z[i]) + Ty_host;
        z_p[i] = scale_host * (R_host[6] * x[i] + R_host[7] * y[i] + R_host[8] * z[i]) + Tz_host;
    }
}

int main() {
    float x[N] = { 4577027.76, 4500468.42, 4488649.48, 4427457.57, 4530092.26,
                  4609646.77, 4633236.71, 4642481.80, 4659521.47, 4670821.34,
                  4611855.60, 4684488.19, 4591391.05, 4747685.29, 4884683.96 };

    float y[N] = { 917648.29, 809968.63, 750093.82, 853754.23, 1005607.52,
                  1109858.03, 1102578.81, 1117776.35, 1263146.4, 1340629.32,
                  1265104.19, 1273453.56, 1120190.62, 1383093.64, 1283099.45 };

    float z[N] = { 4331735.53, 4431508.75, 4453849.34, 4495954.92, 4361162.0,
                  4272977.96, 4228319.66, 4198049.47, 4154320.7, 4117494.83,
                  4206376.25, 4122314.59, 4269079.94, 4014906.72, 3882532.57 };

    float x_p_cpu[N], y_p_cpu[N], z_p_cpu[N];
    float x_p_gpu[N], y_p_gpu[N], z_p_gpu[N];

    // CPU
    auto start_cpu = std::chrono::high_resolution_clock::now();
    transformCPU_parallel(x, y, z, x_p_cpu, y_p_cpu, z_p_cpu);
    auto end_cpu = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> cpu_time = end_cpu - start_cpu;

    // GPU
    float* d_x, * d_y, * d_z, * d_xp, * d_yp, * d_zp;
    hipMalloc(&d_x, N * sizeof(float));
    hipMalloc(&d_y, N * sizeof(float));
    hipMalloc(&d_z, N * sizeof(float));
    hipMalloc(&d_xp, N * sizeof(float));
    hipMalloc(&d_yp, N * sizeof(float));
    hipMalloc(&d_zp, N * sizeof(float));

    hipMemcpy(d_x, x, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_y, y, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_z, z, N * sizeof(float), hipMemcpyHostToDevice);

    auto start_gpu = std::chrono::high_resolution_clock::now();
    transformGPU << <1, N >> > (d_x, d_y, d_z, d_xp, d_yp, d_zp);
    hipDeviceSynchronize();
    auto end_gpu = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> gpu_time = end_gpu - start_gpu;

    hipMemcpy(x_p_gpu, d_xp, N * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(y_p_gpu, d_yp, N * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(z_p_gpu, d_zp, N * sizeof(float), hipMemcpyDeviceToHost);

    // Confronto errori
    int err_count = 0;
    for (int i = 0; i < N; ++i) {
        if (fabs(x_p_cpu[i] - x_p_gpu[i]) > 1e-4 ||
            fabs(y_p_cpu[i] - y_p_gpu[i]) > 1e-4 ||
            fabs(z_p_cpu[i] - z_p_gpu[i]) > 1e-4) {
            err_count++;
        }
    }

    // Info thread e risultati
    int num_threads_cpu = omp_get_max_threads();
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);

    std::cout << "CPU time (parallel): " << cpu_time.count() << " s\n";
    std::cout << "GPU time: " << gpu_time.count() << " s\n";
    std::cout << "Speedup (CPU/GPU): " << cpu_time.count() / gpu_time.count() << "\n";
    std::cout << "Errori CPU/GPU: " << err_count << "\n";
    std::cout << "CPU threads disponibili (OpenMP): " << num_threads_cpu << "\n";
    std::cout << "GPU max threads per block: " << prop.maxThreadsPerBlock << "\n";
    std::cout << "GPU max threads per multiprocessor: " << prop.maxThreadsPerMultiProcessor << "\n";
    std::cout << "GPU number of multiprocessors: " << prop.multiProcessorCount << "\n";

    // Cleanup
    hipFree(d_x); hipFree(d_y); hipFree(d_z);
    hipFree(d_xp); hipFree(d_yp); hipFree(d_zp);

    return 0;
}

